
#include <hip/hip_runtime.h>
#define N (2048 * 2048)
#define THREADS_PER_BLOCK 512

__global__ void add(int *a, int *b, int *c) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  c[index] = a[index] + b[index];
}

void random_ints(int *a, int n){
  int i;
  for(i = 0; i < n; i++){
    a[i] = i;
  }
}

int main(void) {
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof(int);

  //allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  //allocate space for host copies of a, b, c and setup input values
  a = (int *) malloc(size); random_ints(a, N);
  b = (int *) malloc(size); random_ints(b, N);
  c = (int *) malloc(size);

  //copy inputs to device
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

  //Launch add() kernel on GPU with N threads
  add<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c);

  //copy result back to host
  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
  
  //cleanup
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}
