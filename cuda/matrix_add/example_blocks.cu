#include "hip/hip_runtime.h"
#define N 512

__global__ void add(int *a, int *b, int *c) {
  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main(void) {
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof(int);

  //allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  //allocate space for host copies of a, b, c and setup input values
  a = (int *) malloc(size); random_ints(a, N);
  b = (int *) malloc(size); random_ints(b, N);
  c = (int *) malloc(size);

  //copy inputs to device
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

  //Launch add() kernel on GPU with N blocks
  add<<<N,1>>>(d_a, d_b, d_c);

  //copy result back to host
  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
  
  //cleanup
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}
