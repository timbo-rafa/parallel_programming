
#include <hip/hip_runtime.h>
#include<stdio.h>

#define THREADS_PER_BLOCK 1024
#define MEDIUM_THREADS_PER_BLOCK 120
#define SMALL_THREADS_PER_BLOCK 8

__global__ void add(int *a, int *b, int *c, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < n)
    c[index] = a[index] + b[index];

}

void random_ints(int *a, int l, int c){
  int i, j;
  for (i = 0; i < l; ++i){
    for (j = 0; j < c; ++j) {
      a[i * c + j] = i + j;
    }
  }
}

int main(void) {
  int *a, *b, *c, nl, nc;
  int *d_a, *d_b, *d_c;
  int n, blockSize, gridSize, i, size;
  long long int sum;
  scanf("%d %d", &nl, &nc);
  n = nl * nc;
  size = sizeof(int) * n;
  //allocate space for device copies of a, b, c
  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);
  hipMalloc(&d_c, size);

  //allocate space for host copies of a, b, c and setup input values
  a = (int *) malloc(size); random_ints(a, nl, nc);
  b = (int *) malloc(size); random_ints(b, nl, nc);
  c = (int *) malloc(size);

  //copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  if (n < THREADS_PER_BLOCK) {
    if (n < MEDIUM_THREADS_PER_BLOCK) {
      blockSize = SMALL_THREADS_PER_BLOCK;
      if (n < blockSize) blockSize = 1;
    } else {
      blockSize = MEDIUM_THREADS_PER_BLOCK;
    }
  } else {
    blockSize = THREADS_PER_BLOCK;
  }
  gridSize = (int) ceil((float)n/blockSize);

  //Launch add() kernel on GPU with N threads
  add<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

  //copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  sum = 0;
  for(i=0; i < n; i++) {
    sum += c[i];
  }
  printf("%lli\n", sum);
  
  //cleanup
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}
