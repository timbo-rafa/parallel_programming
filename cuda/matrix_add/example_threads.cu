#include "hip/hip_runtime.h"
#define N 512

__global__ void add(int *a, int *b, int *c) {
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main(void) {
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof(int);

  //allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  //allocate space for host copies of a, b, c and setup input values
  a = (int *) malloc(size); random_ints(a, N);
  b = (int *) malloc(size); random_ints(b, N);
  c = (int *) malloc(size);

  //copy inputs to device
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

  //Launch add() kernel on GPU with N threads
  add<<<1, N>>>(d_a, d_b, d_c);

  //copy result back to host
  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
  
  //cleanup
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}
